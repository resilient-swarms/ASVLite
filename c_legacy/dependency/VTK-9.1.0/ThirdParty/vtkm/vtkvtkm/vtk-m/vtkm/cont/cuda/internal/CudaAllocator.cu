//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//============================================================================

#include <cstdlib>
#include <mutex>
#include <vtkm/cont/Logging.h>
#include <vtkm/cont/cuda/ErrorCuda.h>
#include <vtkm/cont/cuda/internal/CudaAllocator.h>
#define NO_VTKM_MANAGED_MEMORY "NO_VTKM_MANAGED_MEMORY"

#include <mutex>
#include <vector>

VTKM_THIRDPARTY_PRE_INCLUDE
#include <hip/hip_runtime.h>
VTKM_THIRDPARTY_POST_INCLUDE

// These static vars are in an anon namespace to work around MSVC linker issues.
namespace
{
#if CUDART_VERSION >= 8000
// Has CudaAllocator::Initialize been called by any thread?
static std::once_flag IsInitialized;
#endif

// Holds how VTK-m currently allocates memory.
// When VTK-m is initialized we set this based on the hardware support ( HardwareSupportsManagedMemory ).
// The user can explicitly disable managed memory through an enviornment variable
// or by calling a function on the CudaAllocator.
// Likewise managed memory can be re-enabled by calling a function on CudaAllocator
// if and only if the underlying hardware supports pageable managed memory
static bool ManagedMemoryEnabled = false;

// True if concurrent pagable managed memory is supported by the machines hardware.
static bool HardwareSupportsManagedMemory = false;

// Avoid overhead of hipMemAdvise and hipMemPrefetchAsync for small buffers.
// This value should be > 0 or else these functions will error out.
static std::size_t Threshold = 1 << 20;
}

namespace vtkm
{
namespace cont
{
namespace cuda
{
namespace internal
{

bool CudaAllocator::UsingManagedMemory()
{
  CudaAllocator::Initialize();
  return ManagedMemoryEnabled;
}

void CudaAllocator::ForceManagedMemoryOff()
{
  if (HardwareSupportsManagedMemory)
  {
    ManagedMemoryEnabled = false;
    VTKM_LOG_F(vtkm::cont::LogLevel::Info, "CudaAllocator disabling managed memory");
  }
  else
  {
    VTKM_LOG_F(
      vtkm::cont::LogLevel::Warn,
      "CudaAllocator trying to disable managed memory on hardware that doesn't support it");
  }
}

void CudaAllocator::ForceManagedMemoryOn()
{
  if (HardwareSupportsManagedMemory)
  {
    ManagedMemoryEnabled = true;
    VTKM_LOG_F(vtkm::cont::LogLevel::Info, "CudaAllocator enabling managed memory");
  }
  else
  {
    VTKM_LOG_F(vtkm::cont::LogLevel::Warn,
               "CudaAllocator trying to enable managed memory on hardware that doesn't support it");
  }
}

bool CudaAllocator::IsDevicePointer(const void* ptr)
{
  CudaAllocator::Initialize();
  if (!ptr)
  {
    return false;
  }

  hipPointerAttribute_t attr;
  hipError_t err = hipPointerGetAttributes(&attr, ptr);
  // This function will return invalid value if the pointer is unknown to the
  // cuda runtime. Manually catch this value since it's not really an error.
  if (err == hipErrorInvalidValue)
  {
    hipGetLastError(); // Clear the error so we don't raise it later...
    return false;
  }
  VTKM_CUDA_CALL(err /*= hipPointerGetAttributes(&attr, ptr)*/);
  return attr.devicePointer == ptr;
}

bool CudaAllocator::IsManagedPointer(const void* ptr)
{
  if (!ptr || !ManagedMemoryEnabled)
  {
    return false;
  }

  hipPointerAttribute_t attr;
  hipError_t err = hipPointerGetAttributes(&attr, ptr);
  // This function will return invalid value if the pointer is unknown to the
  // cuda runtime. Manually catch this value since it's not really an error.
  if (err == hipErrorInvalidValue)
  {
    hipGetLastError(); // Clear the error so we don't raise it later...
    return false;
  }
  VTKM_CUDA_CALL(err /*= hipPointerGetAttributes(&attr, ptr)*/);
#if CUDART_VERSION < 10000 // isManaged deprecated in CUDA 10.
  return attr.isManaged != 0;
#else // attr.type doesn't exist before CUDA 10
  return attr.type == hipMemoryTypeManaged;
#endif
}

void* CudaAllocator::Allocate(std::size_t numBytes)
{
  CudaAllocator::Initialize();
  // When numBytes is zero hipMallocManaged returns an error and the behavior
  // of hipMalloc is not documented. Just return nullptr.
  if (numBytes == 0)
  {
    return nullptr;
  }

  void* ptr = nullptr;
  if (ManagedMemoryEnabled)
  {
    VTKM_CUDA_CALL(hipMallocManaged(&ptr, numBytes));
  }
  else
  {
    VTKM_CUDA_CALL(hipMalloc(&ptr, numBytes));
  }

  {
    VTKM_LOG_F(vtkm::cont::LogLevel::MemExec,
               "Allocated CUDA array of %s at %p.",
               vtkm::cont::GetSizeString(numBytes).c_str(),
               ptr);
  }

  return ptr;
}

void* CudaAllocator::AllocateUnManaged(std::size_t numBytes)
{
  void* ptr = nullptr;
  VTKM_CUDA_CALL(hipMalloc(&ptr, numBytes));
  {
    VTKM_LOG_F(vtkm::cont::LogLevel::MemExec,
               "Allocated CUDA array of %s at %p.",
               vtkm::cont::GetSizeString(numBytes).c_str(),
               ptr);
  }
  return ptr;
}

void CudaAllocator::Free(void* ptr)
{
  VTKM_LOG_F(vtkm::cont::LogLevel::MemExec, "Freeing CUDA allocation at %p.", ptr);
  VTKM_CUDA_CALL(hipFree(ptr));
}

void CudaAllocator::FreeDeferred(void* ptr, std::size_t numBytes)
{
  static std::mutex deferredMutex;
  static std::vector<void*> deferredPointers;
  static std::size_t deferredSize = 0;
  constexpr std::size_t bufferLimit = 2 << 24; //16MB buffer

  {
    VTKM_LOG_F(vtkm::cont::LogLevel::MemExec,
               "Deferring free of CUDA allocation at %p of %s.",
               ptr,
               vtkm::cont::GetSizeString(numBytes).c_str());
  }

  std::vector<void*> toFree;
  // critical section
  {
    std::lock_guard<std::mutex> lock(deferredMutex);
    deferredPointers.push_back(ptr);
    deferredSize += numBytes;
    if (deferredSize >= bufferLimit)
    {
      toFree.swap(deferredPointers);
      deferredSize = 0;
    }
  }

  for (auto&& p : toFree)
  {
    VTKM_LOG_F(vtkm::cont::LogLevel::MemExec, "Freeing deferred CUDA allocation at %p.", p);
    VTKM_CUDA_CALL(hipFree(p));
  }
}

void CudaAllocator::PrepareForControl(const void* ptr, std::size_t numBytes)
{
  if (IsManagedPointer(ptr) && numBytes >= Threshold)
  {
#if CUDART_VERSION >= 8000
    // TODO these hints need to be benchmarked and adjusted once we start
    // sharing the pointers between cont/exec
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetAccessedBy, hipCpuDeviceId));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, hipCpuDeviceId, hipStreamPerThread));
#endif // CUDA >= 8.0
  }
}

void CudaAllocator::PrepareForInput(const void* ptr, std::size_t numBytes)
{
  if (IsManagedPointer(ptr) && numBytes >= Threshold)
  {
#if CUDART_VERSION >= 8000
    int dev;
    VTKM_CUDA_CALL(hipGetDevice(&dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetPreferredLocation, dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetReadMostly, dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetAccessedBy, dev));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, dev, hipStreamPerThread));
#endif // CUDA >= 8.0
  }
}

void CudaAllocator::PrepareForOutput(const void* ptr, std::size_t numBytes)
{
  if (IsManagedPointer(ptr) && numBytes >= Threshold)
  {
#if CUDART_VERSION >= 8000
    int dev;
    VTKM_CUDA_CALL(hipGetDevice(&dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetPreferredLocation, dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseUnsetReadMostly, dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetAccessedBy, dev));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, dev, hipStreamPerThread));
#endif // CUDA >= 8.0
  }
}

void CudaAllocator::PrepareForInPlace(const void* ptr, std::size_t numBytes)
{
  if (IsManagedPointer(ptr) && numBytes >= Threshold)
  {
#if CUDART_VERSION >= 8000
    int dev;
    VTKM_CUDA_CALL(hipGetDevice(&dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetPreferredLocation, dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseUnsetReadMostly, dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetAccessedBy, dev));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, dev, hipStreamPerThread));
#endif // CUDA >= 8.0
  }
}

void CudaAllocator::Initialize()
{
#if CUDART_VERSION >= 8000
  std::call_once(IsInitialized, []() {
    int numDevices;
    VTKM_CUDA_CALL(hipGetDeviceCount(&numDevices));

    if (numDevices == 0)
    {
      return;
    }

    // Check all devices, use the feature set supported by all
    bool managedMemorySupported = true;
    hipDeviceProp_t prop;
    for (int i = 0; i < numDevices && managedMemorySupported; ++i)
    {
      VTKM_CUDA_CALL(hipGetDeviceProperties(&prop, i));
      // We check for concurrentManagedAccess, as devices with only the
      // managedAccess property have extra synchronization requirements.
      managedMemorySupported = managedMemorySupported && prop.concurrentManagedAccess;
    }

    HardwareSupportsManagedMemory = managedMemorySupported;
    ManagedMemoryEnabled = managedMemorySupported;

    VTKM_LOG_F(vtkm::cont::LogLevel::Info,
               "CudaAllocator hardware %s managed memory",
               HardwareSupportsManagedMemory ? "supports" : "doesn't support");

// Check if users want to disable managed memory
#pragma warning(push)
// getenv is not thread safe on windows but since it's inside a call_once block so
// it's fine to suppress the warning here.
#pragma warning(disable : 4996)
    const char* buf = std::getenv(NO_VTKM_MANAGED_MEMORY);
#pragma warning(pop)
    if (managedMemorySupported && buf != nullptr)
    { //only makes sense to disable managed memory if the hardware supports it
      //in the first place
      ManagedMemoryEnabled = false;
      VTKM_LOG_F(
        vtkm::cont::LogLevel::Info,
        "CudaAllocator disabling managed memory due to NO_VTKM_MANAGED_MEMORY env variable");
    }
  });
#endif
}
}
}
}
} // end namespace vtkm::cont::cuda::internal
