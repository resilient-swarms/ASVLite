//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//============================================================================
#include <vtkm/cont/cuda/internal/DeviceAdapterTimerImplementationCuda.h>

#include <vtkm/Types.h>
#include <vtkm/cont/cuda/ErrorCuda.h>

#include <hip/hip_runtime.h>

namespace vtkm
{
namespace cont
{

DeviceAdapterTimerImplementation<
  vtkm::cont::DeviceAdapterTagCuda>::DeviceAdapterTimerImplementation()
{
  VTKM_CUDA_CALL(hipEventCreate(&this->StartEvent));
  VTKM_CUDA_CALL(hipEventCreate(&this->StopEvent));
  this->Reset();
}

DeviceAdapterTimerImplementation<
  vtkm::cont::DeviceAdapterTagCuda>::~DeviceAdapterTimerImplementation()
{
  // These aren't wrapped in VTKM_CUDA_CALL because we can't throw errors
  // from destructors. We're relying on hipGetLastError in the
  // VTKM_CUDA_CHECK_ASYNCHRONOUS_ERROR catching any issues from these calls
  // later.
  hipEventDestroy(this->StartEvent);
  hipEventDestroy(this->StopEvent);
}

void DeviceAdapterTimerImplementation<vtkm::cont::DeviceAdapterTagCuda>::Reset()
{
  this->StartReady = false;
  this->StopReady = false;
}

void DeviceAdapterTimerImplementation<vtkm::cont::DeviceAdapterTagCuda>::Start()
{
  VTKM_CUDA_CALL(hipEventRecord(this->StartEvent, hipStreamPerThread));
  this->StartReady = true;
}

void DeviceAdapterTimerImplementation<vtkm::cont::DeviceAdapterTagCuda>::Stop()
{
  VTKM_CUDA_CALL(hipEventRecord(this->StopEvent, hipStreamPerThread));
  VTKM_CUDA_CALL(hipEventSynchronize(this->StopEvent));
  this->StopReady = true;
}

bool DeviceAdapterTimerImplementation<vtkm::cont::DeviceAdapterTagCuda>::Started() const
{
  return this->StartReady;
}

bool DeviceAdapterTimerImplementation<vtkm::cont::DeviceAdapterTagCuda>::Stopped() const
{
  return this->StopReady;
}

// Callbacks without a mandated order(in independent streams) execute in undefined
// order and maybe serialized. So Instead CudaEventQuery is used here.
// Ref link: https://docs.nvidia.com/cuda/cuda-driver-api/group__CUDA__STREAM.html
bool DeviceAdapterTimerImplementation<vtkm::cont::DeviceAdapterTagCuda>::Ready() const
{
  if (hipEventQuery(this->StopEvent) == hipSuccess)
  {
    return true;
  }
  return false;
}


vtkm::Float64 DeviceAdapterTimerImplementation<vtkm::cont::DeviceAdapterTagCuda>::GetElapsedTime()
  const
{
  assert(this->StartReady);
  if (!this->StartReady)
  {
    VTKM_LOG_F(vtkm::cont::LogLevel::Error,
               "Start() function should be called first then trying to call GetElapsedTime().");
    return 0;
  }
  if (!this->StopReady)
  {
    // Stop was not called, so we have to insert a new event into the stream
    VTKM_CUDA_CALL(hipEventRecord(this->StopEvent, hipStreamPerThread));
    VTKM_CUDA_CALL(hipEventSynchronize(this->StopEvent));
  }

  float elapsedTimeMilliseconds;
  VTKM_CUDA_CALL(hipEventElapsedTime(&elapsedTimeMilliseconds, this->StartEvent, this->StopEvent));
  // Reset Stop flag to its original state
  return static_cast<vtkm::Float64>(0.001f * elapsedTimeMilliseconds);
}
}
} // namespace vtkm::cont
