#include "hip/hip_runtime.h"
//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//============================================================================

#include <vtkm/cont/RuntimeDeviceTracker.h>
#include <vtkm/cont/cuda/DeviceAdapterCuda.h>

#include <vtkm/worklet/DispatcherMapField.h>
#include <vtkm/worklet/WorkletMapField.h>

#include <vtkm/cont/testing/Testing.h>

#include <cmath>
#include <iostream>
#include <limits>
#include <random>
#include <vector>

VTKM_THIRDPARTY_PRE_INCLUDE
#include "hiprand/hiprand_kernel.h"
VTKM_THIRDPARTY_POST_INCLUDE

namespace
{

struct TriggerICE : public vtkm::worklet::WorkletMapField
{
  using ControlSignature = void(FieldIn, FieldIn, FieldOut);
  using ExecutionSignature = _3(_1, _2, WorkIndex);

#ifdef VTKM_CUDA_DEVICE_PASS
  template <class ValueType>
  __device__ ValueType operator()(const ValueType& bad,
                                  const ValueType& sane,
                                  const vtkm::Id sequenceId) const
  {

    hiprandState_t state;
    //Each thread uses same seed but different sequence numbers
    hiprand_init(42, sequenceId, 0, &state);

    int signBad = vtkm::SignBit(bad);
    int signGood = vtkm::SignBit(bad);

    vtkm::Vec<ValueType, 3> coord = { vtkm::Abs(bad * sane),
                                      bad * sane + (ValueType)signBad,
                                      bad * sane + (ValueType)signGood };

    for (int i = 0; i < 10; ++i)
    {
      for (int j = 0; j < 3; ++j)
      {
        if (vtkm::IsNan(coord[j]))
        {
          coord[j] = hiprand_normal(&state) * 5.0f;
          coord[j] = vtkm::Sqrt(vtkm::Dot(coord, coord));
          if (coord[j] <= 1.0f)
          {
            coord[j] += 1.0f;
          }
        }
        if (vtkm::IsInf(coord[j]))
        {
          coord[j] = hiprand_normal(&state) * 8.0f;
          coord[j] = vtkm::Tan(vtkm::Cos(vtkm::Dot(coord, coord)));
        }
      }
    }
    return coord[0] * 4.0f + coord[1] * 4.0f + coord[2] * 4.0f;
  }
#else
  template <class ValueType>
  ValueType operator()(const ValueType& bad, const ValueType& sane, const vtkm::Id sequenceId) const
  {
    return bad + sane * static_cast<ValueType>(sequenceId);
  }
#endif
};

//-----------------------------------------------------------------------------
template <typename Device>
void RunEdgeCases()
{
  std::cout << "Testing complicated worklets that can cause NVCC to ICE." << std::endl;
  //When running CUDA on unsupported hardware we find that IsInf, IsNan, and
  //SignBit can cause the CUDA compiler to crash. This test is a consistent
  //way to detect this.
  //
  //The way it works is we generate all kinds of nasty floating point values
  //such as signaling Nan, quiet Nan, other Nans, +Inf, -Inf, -0, +0, a collection of
  //denormal numbers, and the min and max float values
  //and than a random collection of values from normal float space. We combine this
  //array which we will call 'bad' with another input array which we will call 'sane',
  //We than execute a worklet that takes values stored in 'bad' and 'sane' that does
  //some computation that takes into account the results of IsInf, IsNan, and
  //SignBit
  const vtkm::Id desired_size = 2048;
  std::vector<float> sanevalues;
  std::vector<float> badvalues = { std::numeric_limits<float>::signaling_NaN(),
                                   std::numeric_limits<float>::quiet_NaN(),
                                   std::nanf("1"),
                                   std::nanf("4200042"),
                                   std::numeric_limits<float>::infinity(),
                                   std::numeric_limits<float>::infinity() * -1,
                                   0.0f,
                                   -0.0f,
                                   std::numeric_limits<float>::denorm_min(),
                                   std::nextafter(std::numeric_limits<float>::min(), 0.0f),
                                   std::numeric_limits<float>::denorm_min() *
                                     (1 + std::numeric_limits<float>::epsilon()),
                                   std::nextafter(std::numeric_limits<float>::min(), 0.0f) *
                                     (1 + std::numeric_limits<float>::epsilon()),
                                   std::numeric_limits<float>::lowest(),
                                   std::numeric_limits<float>::min(),
                                   std::numeric_limits<float>::max() };
  const std::size_t bad_size = badvalues.size();
  const vtkm::Id bad_size_as_id = static_cast<vtkm::Id>(bad_size);

  badvalues.reserve(desired_size);
  sanevalues.reserve(desired_size);

  //construct a random number generator
  std::mt19937 rng;
  std::uniform_real_distribution<float> range(-1.0f, 1.0f);

  // now add in some random numbers to the bad values
  for (std::size_t i = 0; i < desired_size - bad_size; ++i)
  {
    badvalues.push_back(range(rng));
  }
  for (std::size_t i = 0; i < desired_size; ++i)
  {
    sanevalues.push_back(range(rng));
  }

  auto bad = vtkm::cont::make_ArrayHandle(badvalues);
  auto sane = vtkm::cont::make_ArrayHandle(sanevalues);
  decltype(sane) result;
  vtkm::worklet::DispatcherMapField<TriggerICE> dispatcher;
  dispatcher.SetDevice(Device());
  dispatcher.Invoke(bad, sane, result);

  auto portal = result.ReadPortal();

  //the first 6 values should be nan
  VTKM_TEST_ASSERT(vtkm::IsNan(portal.Get(0)), "Value should be NaN.");
  VTKM_TEST_ASSERT(vtkm::IsNan(portal.Get(1)), "Value should be NaN.");
  VTKM_TEST_ASSERT(vtkm::IsNan(portal.Get(2)), "Value should be NaN.");
  VTKM_TEST_ASSERT(vtkm::IsNan(portal.Get(3)), "Value should be NaN.");
  VTKM_TEST_ASSERT(vtkm::IsNan(portal.Get(4)), "Value should be NaN.");
  VTKM_TEST_ASSERT(vtkm::IsNan(portal.Get(5)), "Value should be NaN.");

  for (vtkm::Id i = bad_size_as_id; i < desired_size; ++i)
  { //The rest of the values shouldn't be Nan or Inf
    auto v = portal.Get(i);
    const bool valid = !vtkm::IsNan(v) && !vtkm::IsInf(v);
    VTKM_TEST_ASSERT(valid, "value shouldn't be NaN or INF");
  }
}

} //namespace

int UnitTestCudaMathEdgeCases(int argc, char* argv[])
{
  auto& tracker = vtkm::cont::GetRuntimeDeviceTracker();
  tracker.ForceDevice(vtkm::cont::DeviceAdapterTagCuda{});

  return vtkm::cont::testing::Testing::Run(
    RunEdgeCases<vtkm::cont::DeviceAdapterTagCuda>, argc, argv);
}
