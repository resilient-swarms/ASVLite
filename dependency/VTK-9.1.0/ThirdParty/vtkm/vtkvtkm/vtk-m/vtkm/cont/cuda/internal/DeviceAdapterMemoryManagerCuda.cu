//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//============================================================================

#include <vtkm/cont/cuda/DeviceAdapterCuda.h>
#include <vtkm/cont/cuda/ErrorCuda.h>
#include <vtkm/cont/cuda/internal/CudaAllocator.h>
#include <vtkm/cont/cuda/internal/DeviceAdapterMemoryManagerCuda.h>

#include <vtkm/cont/ErrorBadAllocation.h>

#include <vtkm/Math.h>

namespace
{

void* CudaAllocate(vtkm::BufferSizeType size)
{
  try
  {
    return vtkm::cont::cuda::internal::CudaAllocator::Allocate(static_cast<std::size_t>(size));
  }
  catch (const std::exception& error)
  {
    std::ostringstream err;
    err << "Failed to allocate " << size << " bytes on CUDA device: " << error.what();
    throw vtkm::cont::ErrorBadAllocation(err.str());
  }
}

void CudaDelete(void* memory)
{
  if (memory != nullptr)
  {
    vtkm::cont::cuda::internal::CudaAllocator::Free(memory);
  }
};

void CudaReallocate(void*& memory,
                    void*& container,
                    vtkm::BufferSizeType oldSize,
                    vtkm::BufferSizeType newSize)
{
  VTKM_ASSERT(memory == container);

  if (newSize > oldSize)
  {
    // Make a new buffer
    void* newMemory = CudaAllocate(newSize);

    // Copy the data to the new buffer
    VTKM_CUDA_CALL(hipMemcpyAsync(newMemory,
                                   memory,
                                   static_cast<std::size_t>(oldSize),
                                   hipMemcpyDeviceToDevice,
                                   hipStreamPerThread));

    // Reset the buffer in the passed in info
    memory = container = newMemory;
  }
  else
  {
    // Just reuse the buffer.
  }
}

} // anonymous namespace

namespace vtkm
{
namespace cont
{
namespace internal
{

vtkm::cont::internal::BufferInfo DeviceAdapterMemoryManager<
  vtkm::cont::DeviceAdapterTagCuda>::Allocate(vtkm::BufferSizeType size) const
{
  void* memory = CudaAllocate(size);
  return vtkm::cont::internal::BufferInfo(
    vtkm::cont::DeviceAdapterTagCuda{}, memory, memory, size, CudaDelete, CudaReallocate);
}

vtkm::cont::DeviceAdapterId
DeviceAdapterMemoryManager<vtkm::cont::DeviceAdapterTagCuda>::GetDevice() const
{
  return vtkm::cont::DeviceAdapterTagCuda{};
}

vtkm::cont::internal::BufferInfo
DeviceAdapterMemoryManager<vtkm::cont::DeviceAdapterTagCuda>::CopyHostToDevice(
  const vtkm::cont::internal::BufferInfo& src) const
{
  VTKM_ASSERT(src.GetDevice() == vtkm::cont::DeviceAdapterTagUndefined{});

  if (vtkm::cont::cuda::internal::CudaAllocator::IsManagedPointer(src.GetPointer()))
  {
    // In the current code structure, we don't know whether this buffer is going to be used
    // for input or output. (Currently, I don't think there is any difference.)
    vtkm::cont::cuda::internal::CudaAllocator::PrepareForOutput(
      src.GetPointer(), static_cast<std::size_t>(src.GetSize()));

    // The provided control pointer is already cuda managed and can be accessed on the device
    // via unified memory. Just shallow copy the pointer.
    return vtkm::cont::internal::BufferInfo(src, vtkm::cont::DeviceAdapterTagCuda{});
  }
  else
  {
    // Make a new buffer
    vtkm::cont::internal::BufferInfo dest = this->Allocate(src.GetSize());

    this->CopyHostToDevice(src, dest);

    return dest;
  }
}

void DeviceAdapterMemoryManager<vtkm::cont::DeviceAdapterTagCuda>::CopyHostToDevice(
  const vtkm::cont::internal::BufferInfo& src,
  const vtkm::cont::internal::BufferInfo& dest) const
{
  if (vtkm::cont::cuda::internal::CudaAllocator::IsManagedPointer(src.GetPointer()) &&
      src.GetPointer() == dest.GetPointer())
  {
    // In the current code structure, we don't know whether this buffer is going to be used
    // for input or output. (Currently, I don't think there is any difference.)
    vtkm::cont::cuda::internal::CudaAllocator::PrepareForOutput(
      src.GetPointer(), static_cast<std::size_t>(src.GetSize()));

    // The provided pointers are both cuda managed and the same, so the data are already
    // the same.
  }
  else
  {
    vtkm::BufferSizeType size = vtkm::Min(src.GetSize(), dest.GetSize());

    VTKM_LOG_F(vtkm::cont::LogLevel::MemTransfer,
               "Copying host --> CUDA dev: %s (%lld bytes)",
               vtkm::cont::GetHumanReadableSize(static_cast<std::size_t>(size)).c_str(),
               size);

    VTKM_CUDA_CALL(hipMemcpyAsync(dest.GetPointer(),
                                   src.GetPointer(),
                                   static_cast<std::size_t>(size),
                                   hipMemcpyHostToDevice,
                                   hipStreamPerThread));
  }
}


vtkm::cont::internal::BufferInfo
DeviceAdapterMemoryManager<vtkm::cont::DeviceAdapterTagCuda>::CopyDeviceToHost(
  const vtkm::cont::internal::BufferInfo& src) const
{
  VTKM_ASSERT(src.GetDevice() == vtkm::cont::DeviceAdapterTagCuda{});

  vtkm::cont::internal::BufferInfo dest;

  if (vtkm::cont::cuda::internal::CudaAllocator::IsManagedPointer(src.GetPointer()))
  {
    // The provided control pointer is already cuda managed and can be accessed on the host
    // via unified memory. Just shallow copy the pointer.
    vtkm::cont::cuda::internal::CudaAllocator::PrepareForControl(
      src.GetPointer(), static_cast<std::size_t>(src.GetSize()));
    dest = vtkm::cont::internal::BufferInfo(src, vtkm::cont::DeviceAdapterTagUndefined{});

    //In all cases we have possibly multiple async calls queued up in
    //our stream. We need to block on the copy back to control since
    //we don't wanting it accessing memory that hasn't finished
    //being used by the GPU
    vtkm::cont::DeviceAdapterAlgorithm<DeviceAdapterTagCuda>::Synchronize();
  }
  else
  {
    // Make a new buffer
    dest = vtkm::cont::internal::AllocateOnHost(src.GetSize());

    this->CopyDeviceToHost(src, dest);
  }

  return dest;
}

void DeviceAdapterMemoryManager<vtkm::cont::DeviceAdapterTagCuda>::CopyDeviceToHost(
  const vtkm::cont::internal::BufferInfo& src,
  const vtkm::cont::internal::BufferInfo& dest) const
{
  if (vtkm::cont::cuda::internal::CudaAllocator::IsManagedPointer(dest.GetPointer()) &&
      src.GetPointer() == dest.GetPointer())
  {
    // The provided pointers are both cuda managed and the same, so the data are already
    // the same.
  }
  else
  {
    vtkm::BufferSizeType size = vtkm::Min(src.GetSize(), dest.GetSize());

    VTKM_LOG_F(vtkm::cont::LogLevel::MemTransfer,
               "Copying CUDA dev --> host: %s (%lld bytes)",
               vtkm::cont::GetHumanReadableSize(static_cast<std::size_t>(size)).c_str(),
               size);

    VTKM_CUDA_CALL(hipMemcpyAsync(dest.GetPointer(),
                                   src.GetPointer(),
                                   static_cast<std::size_t>(size),
                                   hipMemcpyDeviceToHost,
                                   hipStreamPerThread));
  }

  //In all cases we have possibly multiple async calls queued up in
  //our stream. We need to block on the copy back to control since
  //we don't wanting it accessing memory that hasn't finished
  //being used by the GPU
  vtkm::cont::DeviceAdapterAlgorithm<DeviceAdapterTagCuda>::Synchronize();
}

vtkm::cont::internal::BufferInfo
DeviceAdapterMemoryManager<vtkm::cont::DeviceAdapterTagCuda>::CopyDeviceToDevice(
  const vtkm::cont::internal::BufferInfo& src) const
{
  vtkm::cont::internal::BufferInfo dest = this->Allocate(src.GetSize());
  this->CopyDeviceToDevice(src, dest);

  return dest;
}

void DeviceAdapterMemoryManager<vtkm::cont::DeviceAdapterTagCuda>::CopyDeviceToDevice(
  const vtkm::cont::internal::BufferInfo& src,
  const vtkm::cont::internal::BufferInfo& dest) const
{
  VTKM_CUDA_CALL(hipMemcpyAsync(dest.GetPointer(),
                                 src.GetPointer(),
                                 static_cast<std::size_t>(src.GetSize()),
                                 hipMemcpyDeviceToDevice,
                                 hipStreamPerThread));
}
}
}
} // namespace vtkm::cont::internal
